#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/sigmoid_cross_entropy_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

#ifdef USE_GREENTEA
#include "caffe/greentea/greentea.hpp"
#include "caffe/greentea/greentea_math_functions.hpp"
#endif

namespace caffe {

#ifdef USE_CUDA
template<typename Dtype>
__global__ void SigmoidCrossEntropyLossForwardGPU(
                    const int_tp nthreads,
                    const Dtype* input_data,
                    const Dtype* target, Dtype* loss ) {
    CUDA_KERNEL_LOOP(i, nthreads) {
        loss[i] = -(input_data[i]*(target[i] - (input_data[i] >= 0)) -
            log(1 + exp(input_data[i] - 2*input_data[i]*(input_data[i] >= 0))));
    }
}
#endif  // USE_CUDA

// Forward GPU
template<typename Dtype>
void SigmoidCrossEntropyLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
    // The forward pass computes the sigmoid outputs.
    sigmoid_bottom_vec_[0] = bottom[0];
    sigmoid_layer_->Forward(sigmoid_bottom_vec_, sigmoid_top_vec_);

    if (this->device_->backend() == BACKEND_CUDA) {

        const int_tp count = bottom[0]->count();
        const int_tp num = bottom[0]->num();

        // Stable version of loss computation from input data
        const Dtype* input_data = bottom[0]->gpu_data();
        const Dtype* target = bottom[1]->gpu_data();

        // Since this memory is not used for anything until it is overwritten
        // on the backward pass, we use it here to avoid having to allocate new
        // GPU memory to accumulate intermediate results in the kernel.
        Dtype* loss_data = bottom[0]->mutable_gpu_diff();

        SigmoidCrossEntropyLossForwardGPU<Dtype> CUDA_KERNEL(
                CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS)(
            count, input_data, target, loss_data);
        CUDA_POST_KERNEL_CHECK;

        Dtype loss;
        caffe_gpu_asum(count, loss_data, &loss);
        top[0]->mutable_cpu_data()[0] = loss / num;
    }
}

template<typename Dtype>
void SigmoidCrossEntropyLossLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL)<< this->type()
    << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    const int_tp count = bottom[0]->count();
    const int_tp num = bottom[0]->num();
    const Dtype* sigmoid_output_data = sigmoid_output_->gpu_data();
    const Dtype* target = bottom[1]->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

    if (this->device_->backend() == BACKEND_CUDA) {
#ifdef USE_CUDA
      // First, compute the diff
      caffe_copy(count, sigmoid_output_data, bottom_diff);
      caffe_gpu_axpy(count, Dtype(-1), target, bottom_diff);
      // Scale down gradient
      const Dtype loss_weight = top[0]->cpu_diff()[0];
      caffe_gpu_scal(count, loss_weight / num, bottom_diff);
#endif  // USE_CUDA
    } else {
#ifdef USE_GREENTEA
      viennacl::ocl::context &ctx = viennacl::ocl::get_context(
          this->device_->id());

      // First, compute the diff
      greentea_copy<Dtype>(count, (cl_mem)sigmoid_output_data, 0,
                           (cl_mem)bottom_diff, 0, &ctx);
      greentea_gpu_axpy<Dtype>(this->device_->id(), count,
                               Dtype(-1), (cl_mem)target, 0,
                               (cl_mem)bottom_diff, 0);
      // Scale down gradient
      const Dtype loss_weight = top[0]->cpu_diff()[0];
      greentea_gpu_scal(this->device_->id(), count, loss_weight / num,
                        (cl_mem)bottom_diff, 0);
#endif  // USE_GREENTEA
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SigmoidCrossEntropyLossLayer);

}  // namespace caffe
